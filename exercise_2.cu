
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double mysecond(){
  struct timeval tp;
  struct timezone tzp;
  int i = gettimeofday(&tp, &tzp);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void SAXPY_CPU(int N, float A, float *X, float *Y, float *R){
  for(int i=0; i<N; i++){
    R[i] = A * X[i] + Y[i];
  }
}

__global__ void SAXPY_GPU(float A, float *X, float *Y){
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  Y[i] = A * X[i] + Y[i];
}

int main(int argc, char *argv[]){

  int ARRAY_SIZE = 10000;
  if(argc>0) ARRAY_SIZE = atoi(argv[1]);
  printf("ARRAY_SIZE: %d\n", ARRAY_SIZE);

  float A = 10;
  float *X_CPU = (float *) malloc(ARRAY_SIZE*sizeof(float));
  float *Y_CPU = (float *) malloc(ARRAY_SIZE*sizeof(float));
  float *R_CPU = (float *) malloc(ARRAY_SIZE*sizeof(float));
  for(int i=0; i<ARRAY_SIZE; i++){
    X_CPU[i] = float(i)/ARRAY_SIZE * 2;
    Y_CPU[i] = float(i)/ARRAY_SIZE * 4;
  }

  // CPU Part:
  double T_CPU = mysecond();
  SAXPY_CPU(ARRAY_SIZE, A, X_CPU, Y_CPU, R_CPU);
  T_CPU = mysecond() - T_CPU;
  printf("Computing SAXPY on the CPU... Done! Time: %f\n", T_CPU);

  // GPU Part:
  float *X_GPU = 0;
  float *Y_GPU = 0;
  float *R_GPU = (float *) malloc(ARRAY_SIZE*sizeof(float));
  hipMalloc(&X_GPU, ARRAY_SIZE*sizeof(float));
  hipMalloc(&Y_GPU, ARRAY_SIZE*sizeof(float));
  hipMemcpy(X_GPU, X_CPU, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Y_GPU, Y_CPU, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  int TPB = 256;
  int BPG = (ARRAY_SIZE+TPB-1)/TPB;
  double T_GPU = mysecond();
  SAXPY_GPU<<<BPG, TPB>>>(A, X_GPU, Y_GPU);
  hipMemcpy(R_GPU, Y_GPU, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  T_GPU = mysecond() - T_GPU;
  printf("Computing SAXPY on the GPU... Done! Time: %f\n", T_GPU);


  // Comparison Part:
  float maxError = -10.0;
  for(int i=0; i<ARRAY_SIZE; i++){
    maxError = fmax(maxError, fabs(R_CPU[i]-R_GPU[i]));
  }
  if(maxError<0.00001){
   printf("Comparing the output for each implementation... Correct! Max Error: %e\n", maxError);
  }
  else{
   printf("Not Correct! Max Error: %e\n", maxError);
  }

  //printf("%d, %f, %f, %e\n", ARRAY_SIZE, T_CPU, T_GPU, maxError);

  free(X_CPU);
  free(Y_CPU);
  hipFree(X_GPU);
  hipFree(Y_GPU);
  return 0;

}


